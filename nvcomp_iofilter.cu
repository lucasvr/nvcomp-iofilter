#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hdf5.h>

#include <nvcomp/snappy.hpp>
#include <nvcomp.hpp>

#include "nvcomp_iofilter.h"

using namespace nvcomp;

bool compress(void **buf, size_t *buf_size, size_t *compressed_size)
{
    bool retval = true;

    // Copy input data to device memory
    uint8_t *uncomp_buf;
    hipMalloc(&uncomp_buf, *buf_size);
    hipMemcpy(uncomp_buf, *buf, *buf_size, hipMemcpyDefault);

    // Create stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Configure nvCOMP manager
    const int chunk_size = 1 << 16;
    SnappyManager nvcomp_manager{chunk_size, stream};
    CompressionConfig comp_config = nvcomp_manager.configure_compression(*buf_size);

    // Compress
    uint8_t *comp_buffer;
    hipMalloc(&comp_buffer, comp_config.max_compressed_buffer_size);
    nvcomp_manager.compress(uncomp_buf, comp_buffer, comp_config);
    size_t output_size = nvcomp_manager.get_compressed_output_size(comp_buffer);
    hipStreamSynchronize(stream);

    // Handle the unexpected case in which the output buffer is too small
    // (i.e., we must be compressing random data)
    if (*buf_size < output_size) {
        char *newbuf = (char *) malloc(sizeof(char) * output_size);
        if (! newbuf) {
            fprintf(stderr, "Not enough memory to hold the compressed data\n");
            retval = false;
        } else {
            free(*buf);
            *buf = newbuf;
            *buf_size = output_size;
        }
    }

    // Replace input data with output data
    if (retval) {
        hipMemcpy(*buf, comp_buffer, output_size, hipMemcpyDefault);
        *compressed_size = output_size;
    }

    hipFree(comp_buffer);
    hipFree(uncomp_buf);
    hipStreamDestroy(stream);
    return retval;
}

bool uncompress(void **buf, size_t *buf_size, size_t *uncompressed_size)
{
    // Copy input data to device memory
    uint8_t *comp_buffer;
    hipMalloc(&comp_buffer, *buf_size);
    hipMemcpy(comp_buffer, *buf, *buf_size, hipMemcpyDefault);

    // Create stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // Configure nvCOMP manager
    const int chunk_size = 1 << 16;
    SnappyManager nvcomp_manager{chunk_size, stream};
    DecompressionConfig decomp_config = nvcomp_manager.configure_decompression(comp_buffer);
    size_t output_size = decomp_config.decomp_data_size;

    // Uncompress
    uint8_t *uncomp_buffer;
    hipMalloc(&uncomp_buffer, output_size);
    nvcomp_manager.decompress(uncomp_buffer, comp_buffer, decomp_config);
    hipStreamSynchronize(stream);

    // Replace input with output data
    if (*buf_size < output_size) {
        // The buffer provided by HDF5 is not large enough to hold the uncompressed data
        char *newbuf = (char *) malloc(sizeof(char) * output_size);
        if (! newbuf) {
            fprintf(stderr, "Not enough memory to hold the uncompressed data\n");
            return false;
        }
        free(*buf);
        *buf = newbuf;
        *buf_size = output_size;
    }
    hipMemcpy(*buf, uncomp_buffer, output_size, hipMemcpyDefault);
    *uncompressed_size = output_size;

    hipFree(uncomp_buffer);
    hipFree(comp_buffer);
    hipStreamDestroy(stream);
    return true;
}

static size_t filter_callback(unsigned int flags, size_t cd_nelmts,
    const unsigned int *cd_values, size_t nbytes, size_t *buf_size, void **buf)
{
    size_t output_size = 0;

    if (flags & H5Z_FLAG_REVERSE) {
        if (uncompress(buf, buf_size, &output_size) == false)
            return 0;
    } else {
        if (compress(buf, buf_size, &output_size) == false)
            return 0;
    }

    return output_size;
}

extern "C" const H5Z_class2_t NVCOMP_FILTER[1] = {{
    H5Z_CLASS_T_VERS,
    NVCOMP_FILTER_ID,
    1, 1,
    "nvcomp_filter",
    NULL, /* can_apply */
    NULL, /* set_local */
    filter_callback,
}};

extern "C" H5PL_type_t H5PLget_plugin_type(void) { return H5PL_TYPE_FILTER; }
extern "C" const void *H5PLget_plugin_info(void) { return NVCOMP_FILTER; }
